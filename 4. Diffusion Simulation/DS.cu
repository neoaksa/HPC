/* Diffusion Simulation 
 * nvcc -arch=sm_30 DS.cu -run
 * */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <sys/time.h>
// set a 3D volume
//define the data set size (cubic volume)
#define DATAXSIZE 64
#define DATAYSIZE 64
#define DATAZSIZE 64
//block size = 8*8*8 = 512
#define BLKXSIZE 8
#define BLKYSIZE 8
#define BLKZSIZE 8
//time iteration
#define t 10000


// device function to set the 3D volume
__global__ void diffusion(float (*output_array)[DATAYSIZE][DATAXSIZE],
                          float (*shadow_array)[DATAYSIZE][DATAXSIZE])
{   
//     // get grid, only works on GTX 1000 up
//     grid_group g = this_grid();
    // get position
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int idy = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int idz = blockIdx.z*blockDim.z + threadIdx.z;
    
    // not the edge
    if(idx>0 && idx<DATAXSIZE-1 && idy>0 && idy<DATAYSIZE-1 && idz>0 && idz<DATAZSIZE-1){
        output_array[idz][idy][idx] = (shadow_array[idz][idy][idx-1] + shadow_array[idz][idy][idx+1]
                                    + shadow_array[idz][idy-1][idx] + shadow_array[idz][idy+1][idx]
                                    + shadow_array[idz-1][idy][idx] + shadow_array[idz+1][idy][idx])/6;
    }
    // reach to the edge to rebound
    else{
        int nbr = 6;
        if(idx==0 || idx==DATAXSIZE-1) nbr-=1;
        if(idy==0 || idy==DATAYSIZE-1) nbr-=1;
        if(idz==0 || idz==DATAZSIZE-1) nbr-=1;
        output_array[idz][idy][idx] = (((idx==0)? 0:shadow_array[idz][idy][idx-1]) + ((idx==(DATAXSIZE-1))? 0: shadow_array[idz][idy][idx+1])
                                    + ((idy==0)? 0:shadow_array[idz][idy-1][idx]) + ((idy==(DATAYSIZE-1))? 0: shadow_array[idz][idy+1][idx])
                                    + ((idz==0)? 0:shadow_array[idz-1][idy][idx])+ ((idz==(DATAZSIZE-1))? 0: shadow_array[idz+1][idy][idx]))/nbr;
    }
}

// refresh shadow array
__global__ void refesh(float (*output_array)[DATAYSIZE][DATAXSIZE],
                          float (*shadow_array)[DATAYSIZE][DATAXSIZE])
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int idy = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int idz = blockIdx.z*blockDim.z + threadIdx.z;
    // updae shadow and reset barrier
    shadow_array[idz][idy][idx] = output_array[idz][idy][idx];
}

// cpu version for validation
void diffusion_cpu(float (*output_array)[DATAYSIZE][DATAXSIZE],
                   float (*shadow_array)[DATAYSIZE][DATAXSIZE])
{   
    for(int time=1; time<=t; time++){
        for(int idz=0;idz<DATAZSIZE;idz++)
            for(int idy=0;idy<DATAYSIZE;idy++)
                for(int idx=0;idx<DATAXSIZE;idx++){
                    if(idx>0 && idx<DATAXSIZE-1 && idy>0 && idy<DATAYSIZE-1 && idz>0 && idz<DATAZSIZE-1){
                        output_array[idz][idy][idx] = (shadow_array[idz][idy][idx-1] + shadow_array[idz][idy][idx+1]
                                                    + shadow_array[idz][idy-1][idx] + shadow_array[idz][idy+1][idx]
                                                    + shadow_array[idz-1][idy][idx] + shadow_array[idz+1][idy][idx])/6;
                    }
                    // reach to the edge to rebound
                    else{
                        int nbr = 6;
                        if(idx==0 || idx==DATAXSIZE-1) nbr-=1;
                        if(idy==0 || idy==DATAYSIZE-1) nbr-=1;
                        if(idz==0 || idz==DATAZSIZE-1) nbr-=1;
                        output_array[idz][idy][idx] = (((idx==0)? 0:shadow_array[idz][idy][idx-1]) + ((idx==(DATAXSIZE-1))? 0: shadow_array[idz][idy][idx+1])
                                                    + ((idy==0)? 0:shadow_array[idz][idy-1][idx]) + ((idy==(DATAYSIZE-1))? 0: shadow_array[idz][idy+1][idx])
                                                    + ((idz==0)? 0:shadow_array[idz-1][idy][idx])+ ((idz==(DATAZSIZE-1))? 0: shadow_array[idz+1][idy][idx]))/nbr;
//                         printf("%d,%d,%d-%f \n", idz,idy,idx, output_array[idz][idy][idx]);
                    }
                }
        // updae shadow and reset barrier/signal
        for(int idz=0;idz<DATAZSIZE;idz++)
            for(int idy=0;idy<DATAYSIZE;idy++)
                for(int idx=0;idx<DATAXSIZE;idx++){
                    shadow_array[idz][idy][idx] = output_array[idz][idy][idx];
                }
    }
}

int main(int argc, char *argv[])
{
    typedef float nRarray[DATAYSIZE][DATAXSIZE];
    // overall data set sizes
    const int nx = DATAXSIZE;
    const int ny = DATAYSIZE;
    const int nz = DATAZSIZE;
    // error code
    hipError_t result;
    // initial position
    unsigned int init_pos[6] = {0};
    init_pos[0] 	= DATAZSIZE/2 - 2;// x min
    init_pos[1] 	= DATAZSIZE/2 + 2;// x max
    init_pos[2] 	= DATAYSIZE/2 - 2;// y min
    init_pos[3] 	= DATAYSIZE/2 + 2;// y max
    init_pos[4] 	= DATAXSIZE/2 - 2;// z min
    init_pos[5] 	= DATAXSIZE/2 + 2;// z max
    //initial concetration
    const float con_begin = 20000.0;
    
    // pointers for data set storage via malloc
    nRarray *output_c; // storage for result stored on host
    nRarray *output_d; // storage for result computed on device
    nRarray *shadow_c; // shadow array for saving temp value on host
    nRarray *shadow_d; // shadow array for saving temp value
    nRarray *output_cpu; // for cpu version
    nRarray *shadow_cpu; // for cpu version
    // allocate storage for receiving output
    if ((output_c = (nRarray *)malloc((nx*ny*nz)*sizeof(float))) == 0) {
		fprintf(stderr,"malloc1 Fail \n"); return 1;
	}
	// allocate storage for shadow arry
    if ((shadow_c = (nRarray *)malloc((nx*ny*nz)*sizeof(float))) == 0) {
		fprintf(stderr,"malloc1 Fail \n"); return 1;
	}
    if ((output_cpu = (nRarray *)malloc((nx*ny*nz)*sizeof(float))) == 0) {
		fprintf(stderr,"malloc1 Fail \n"); return 1;
	}
	// allocate storage for shadow arry
    if ((shadow_cpu = (nRarray *)malloc((nx*ny*nz)*sizeof(float))) == 0) {
		fprintf(stderr,"malloc1 Fail \n"); return 1;
	}
    // inital concetration
    for(int k=init_pos[0];k<=init_pos[1];k++)
        for(int j=init_pos[2]; j<=init_pos[3];j++) 
            for(int i=init_pos[4];i<=init_pos[5];i++ ){
                output_c[k][j][i] = con_begin;
                shadow_c[k][j][i] = con_begin;
                output_cpu[k][j][i] = con_begin;
                shadow_cpu[k][j][i] = con_begin;
    }
    // allocate GPU device buffers
    result = hipMalloc((void **) &output_d, (nx*ny*nz)*sizeof(float));
    if (result != hipSuccess) {
		fprintf(stderr, ("Failed to allocate device buffer-output_d"));
		exit(1);
	}
    result = hipMalloc((void **) &shadow_d, (nx*ny*nz)*sizeof(float));
    if (result != hipSuccess) {
		fprintf(stderr, ("Failed to allocate device buffer- shadow_d"));
		exit(1);
	}
	// copy host to device
	result = hipMemcpy(shadow_d, shadow_c, (nx*ny*nz)*sizeof(float), hipMemcpyHostToDevice);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMemcpy host->dev (block) failed.");
		exit(1);
	}
    result = hipMemcpy(output_d, output_c, (nx*ny*nz)*sizeof(float), hipMemcpyHostToDevice);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMemcpy host->dev (block) failed.");
		exit(1);
	}
	
    //timing 
	struct timeval start_cpu, finish_cpu,start_gpu, finish_gpu;
	// timing start
	gettimeofday (&start_gpu, NULL);
    
    // compute result
    const dim3 blockSize(BLKXSIZE, BLKYSIZE, BLKZSIZE);
    const dim3 gridSize((DATAXSIZE/BLKXSIZE), (DATAYSIZE/BLKYSIZE), (DATAZSIZE/BLKZSIZE));
    // loop with time t
    for(int time=1; time<=t; time++){
        diffusion<<<gridSize,blockSize>>>(output_d,shadow_d);
        refesh<<<gridSize,blockSize>>>(output_d,shadow_d);
//     	cudaDeviceSynchronize();
    }
    // copy output data back to host
    result = hipMemcpy(output_c, output_d, ((nx*ny*nz)*sizeof(float)), hipMemcpyDeviceToHost);
    if (result != hipSuccess) {
		fprintf(stderr, ("hipMemcpy dev (block)->host failed."));
		exit(1);
	}
    //timing end
	gettimeofday (&finish_gpu, NULL);
	double elapsed_gpu = (finish_gpu.tv_sec - start_gpu.tv_sec)*1000000 + finish_gpu.tv_usec - start_gpu.tv_usec;
    
    // timing start
	gettimeofday (&start_cpu, NULL);
    //cpu version
    diffusion_cpu(output_cpu,shadow_cpu);
    //timing end
	gettimeofday (&finish_cpu, NULL);
	double elapsed_cpu = (finish_cpu.tv_sec - start_cpu.tv_sec)*1000000 + finish_cpu.tv_usec - start_cpu.tv_usec;   
    
    //check two version
    for (unsigned i=0; i<nz; i++)
      for (unsigned j=0; j<ny; j++)
        for (unsigned k=0; k<nx; k++){
            if(output_c[i][j][k]!=output_cpu[i][j][k])
            {
                printf("check error happen \n");
                printf("position:[%d][%d][%d]. value:%f:%f \n",i,j,k ,output_c[i][j][k],output_cpu[i][j][k]);
//                 return 0;
                
            }
        }
    
    	
	printf("Time spent(GPU) of dt = %d: %f \n",t,elapsed_gpu);	
	printf("Time spent(CPU) of dt = %d: %f \n",t,elapsed_cpu);

    //write result to file
	std::ofstream myfile;
	myfile.open ("DS-2.csv",std::ios_base::app);
//     myfile.open ("DS.csv");
    for (unsigned i=0; i<nz; i++)
      for (unsigned j=0; j<ny; j++)
        for (unsigned k=0; k<nx; k++){
            if( output_c[i][j][k]!=0)
            myfile << i << "," << j << "," << k << "," << output_c[i][j][k] << "," << std::to_string(t) << std::endl;
        }
	myfile.close();
	// free memory
    free(output_c);
    free(shadow_c);
    result = hipFree(shadow_d);
    if (result != hipSuccess) {
		fprintf(stderr, ("Failed to Free device buffer - shadow_d"));
		exit(1);
	}
    result = hipFree(output_d);
    if (result != hipSuccess) {
		fprintf(stderr, ("Failed to Free device buffer - output_d"));
		exit(1);
	}

    return 0;
}


