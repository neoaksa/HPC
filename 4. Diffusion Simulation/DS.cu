/* Diffusion Simulation 
 * nvcc -arch=sm_30 DS.cu -run
 * To compile with: nvcc -O2 -o DS DS.cu
 * reference:Inter-Block GPU Communication via Fast Barrier Synchronization
 * */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>


// set a 3D volume
//define the data set size (cubic volume)
#define DATAXSIZE 512
#define DATAYSIZE 512
#define DATAZSIZE 512
//block size = 8*8*8 = 512
#define BLKXSIZE 8
#define BLKYSIZE 8
#define BLKZSIZE 8
//time iteration
#define t 10

__device__ int barrier = DATAXSIZE/BLKXSIZE;

// device function to set the 3D volume
__global__ void diffusion(unsigned int *init_pos, float (*output_array)[DATAYSIZE][DATAXSIZE],
                          float (*shadow_array)[DATAYSIZE][DATAXSIZE], float con_begin)
{   
//     // get grid, only works on new GPU
//     grid_group g = this_grid();
    // get position
    unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned idy = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned idz = blockIdx.z*blockDim.z + threadIdx.z;
    // inital concetration
    if(idx>=init_pos[0] && idx<=init_pos[1] 
        && idy>=init_pos[2] && idy<=init_pos[3] 
        && idz>=init_pos[4] && idz<=init_pos[5] ){
            output_array[idz][idy][idx] = con_begin;
            shadow_array[idz][idy][idx] = con_begin;
        }
     __syncthreads();
    // diffusion in 3D space 
    // U(t+1) equal to avg of U(t) of six neigbors
    for(int i=1;i<=t;i++){
        //refresh array
        if(idx>0 && idx<DATAXSIZE-1 && idy>0 && idy<DATAYSIZE-1 && idz>0 && idz<DATAZSIZE-1){
            output_array[idz][idy][idx] = (shadow_array[idz][idy][idx-1] + shadow_array[idz][idy][idx+1]
                                        + shadow_array[idz][idy-1][idx] + shadow_array[idz][idy+1][idx]
                                        + shadow_array[idz-1][idy][idx] + shadow_array[idz+1][idy][idx])/6;
        }
//         //sync within grid
//         g.sync();
        //sync between blocks
         __syncthreads();
        if ( threadIdx.x == 0 && threadIdx.y==0 && threadIdx.z ==0 )
            atomicSub( &barrier , 1 );
        
        /* Now wait for the barrier to be zero. */
        if ( threadIdx.x == 0 && threadIdx.y==0 && threadIdx.z ==0  ){
            while ( atomicCAS( &barrier , 0 , 0 ) != 0 );
        }
//         // refresh shadow array
//         shadow_array[idz][idy][idx] = output_array[idz][idy][idx];
//         /* Make sure everybody has waited for the barrier. */
        __syncthreads();
//          if ( threadIdx.x == 0 && threadIdx.y==0 && threadIdx.z ==0  && atomicCAS( &barrier , 0 , 0 ) == 0 ){
//             atomicAdd(&barrier, DATAXSIZE/BLKXSIZE);
//          }
//         /* Make sure everybody has waited for the barrier. */
//         __syncthreads();
    }
}

int main(int argc, char *argv[])
{
    typedef float nRarray[DATAYSIZE][DATAXSIZE];
    // overall data set sizes
    const int nx = DATAXSIZE;
    const int ny = DATAYSIZE;
    const int nz = DATAZSIZE;
    // error code
    hipError_t result;
    // initial position
    unsigned int init_position[6] = {0};
    init_position[0] 	= DATAXSIZE/2 - 2;// x min
    init_position[1] 	= DATAXSIZE/2 + 2;// x max
    init_position[2] 	= DATAYSIZE/2 - 2;// y min
    init_position[3] 	= DATAYSIZE/2 + 2;// y max
    init_position[4] 	= DATAZSIZE/2 - 2;// z min
    init_position[5] 	= DATAZSIZE/2 + 2;// z max
    //initial concetration
    const float con_begin = 200.0;
    // pointers for data set storage via malloc
    unsigned int *inital_d; // initial the start status in device
    nRarray *output_c; // storage for result stored on host
    nRarray *output_d;  // storage for result computed on device
    nRarray *shadow_d; // shadow array for saving temp value
    // allocate storage for receiving output
    if ((output_c = (nRarray *)malloc((nx*ny*nz)*sizeof(float))) == 0) {
		fprintf(stderr,"malloc1 Fail \n"); return 1;
	}
    // allocate GPU device buffers
	result = hipMalloc((void **) &inital_d, (6)*sizeof(unsigned int));
    if (result != hipSuccess) {
		fprintf(stderr, ("Failed to allocate device buffer-inital_d"));
		exit(1);
	}
    result = hipMalloc((void **) &output_d, (nx*ny*nz)*sizeof(float));
    if (result != hipSuccess) {
		fprintf(stderr, ("Failed to allocate device buffer-output_d"));
		exit(1);
	}
    result = hipMalloc((void **) &shadow_d, (nx*ny*nz)*sizeof(float));
    if (result != hipSuccess) {
		fprintf(stderr, ("Failed to allocate device buffer- shadow_d"));
		exit(1);
	}
	// copy host to device
	result = hipMemcpy(inital_d, init_position, ((6)*sizeof(float)), hipMemcpyHostToDevice);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMemcpy host->dev (block) failed.");
		exit(1);
	}
    // compute result
    const dim3 blockSize(BLKXSIZE, BLKYSIZE, BLKZSIZE);
    const dim3 gridSize((DATAXSIZE/BLKXSIZE), (DATAYSIZE/BLKYSIZE), (DATAZSIZE/BLKZSIZE));
    
    diffusion<<<gridSize,blockSize>>>(inital_d,output_d,shadow_d,con_begin);
    // copy output data back to host
    result = hipMemcpy(output_c, output_d, ((nx*ny*nz)*sizeof(float)), hipMemcpyDeviceToHost);
    if (result != hipSuccess) {
		fprintf(stderr, ("hipMemcpy dev (block)->host failed."));
		exit(1);
	}
	
	//TEST
    for (unsigned i=0; i<nz; i++)
      for (unsigned j=0; j<ny; j++)
        for (unsigned k=0; k<nx; k++){
            if(output_c[i][j][k]!=0)
            printf("%d-%f ",i+j+k,output_c[i][j][k]);
        }
	
	// free memory
    free(output_c);
    result = hipFree(shadow_d);
    if (result != hipSuccess) {
		fprintf(stderr, ("Failed to allocate device buffer"));
		exit(1);
	}
    result = hipFree(output_d);
    if (result != hipSuccess) {
		fprintf(stderr, ("Failed to allocate device buffer"));
		exit(1);
	}
	result = hipFree(inital_d);
    if (result != hipSuccess) {
		fprintf(stderr, ("Failed to allocate device buffer"));
		exit(1);
	}
    return 0;
}
